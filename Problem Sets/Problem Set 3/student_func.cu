#include "hip/hip_runtime.h"
#include "utils.h"

__global__ void calc(const float* const d_in, int *d_bins, unsigned int* const d_cdf, float* d_min, float* d_max, size_t numRows, size_t numCols, size_t numBins)
{
   size_t threadPos = threadIdx.x + blockDim.x * blockIdx.x;
   float curIn = d_in[threadPos];
   //step 1
   while(*d_min > curIn)
   {
      *d_min = curIn;
      __threadfence();
   }
   while(*d_max < curIn)
   {
      *d_max = curIn;
      __threadfence();
   }
   float curMin = *d_min, curMax = *d_max;
   //atomicMin(&d_min, curIn);
   //atomicMax(&d_max, curIn);
   //__syncthreads();
   //step 2
   
   float logLumRange = curMax - curMin;
   //step 3
   size_t bin = min((unsigned int)(numBins - 1), (unsigned int)((curIn - curMin) / logLumRange * numBins));
   atomicAdd(&d_bins[bin], 1);
   __syncthreads();
   
   //step 4
   if (0 == threadPos || threadPos >= numBins)
      return;
   d_cdf[threadPos] = d_cdf[threadPos - 1] + d_bins[threadPos - 1];
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
   int* d_bins;
   float* d_min, *d_max;
   checkCudaErrors(hipMalloc(&d_bins,   sizeof(int) * numBins));
   checkCudaErrors(hipMemset(d_bins, 0,   sizeof(int) * numBins));
   checkCudaErrors(hipMalloc(&d_min,   sizeof(float)));
   checkCudaErrors(hipMemset(d_min, 0,   sizeof(float)));
   checkCudaErrors(hipMalloc(&d_max,   sizeof(float)));
   checkCudaErrors(hipMemset(d_max, 0,   sizeof(int)));
   
   const size_t length = numRows * numCols;
   calc<<<numBins, length/numBins>>>(d_logLuminance, d_bins, d_cdf, d_min, d_max, numRows, numCols, numBins);
   checkCudaErrors(hipGetLastError());
   
   hipMemcpy(d_min, &min_logLum, sizeof(float), hipMemcpyDeviceToHost);
   hipMemcpy(d_max, &max_logLum, sizeof(float), hipMemcpyDeviceToHost);
   
   hipFree(d_bins);
   hipFree(d_min);
   hipFree(d_max);
}
