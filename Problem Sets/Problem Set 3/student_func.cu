#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>

__global__ void calc(const float* const d_in, int *d_bins, unsigned int* const d_cdf, float* d_min, float* d_max, size_t numRows, size_t numCols, size_t numBins)
{
   size_t threadPos = threadIdx.x + blockDim.x * blockIdx.x;
   float curIn = d_in[threadPos];
   
   //step 1
   while(*d_min > curIn)
   {
      *d_min = curIn;
      __threadfence();
   }
   while(*d_max < curIn)
   {
      *d_max = curIn;
      __threadfence();
   }
   __syncthreads();
   float curMin = *d_min, curMax = *d_max;
   
   //step 2
   int result = ((curIn - curMin) / (curMax - curMin)) * (float)numBins;
   
   //step 3
   int bin = min((int)numBins - 1, result);
   atomicAdd(&d_bins[bin], 1);
   __syncthreads();
  
   //step 4
   if (0 == threadPos || threadPos > numBins - 1)
      return;

   //scan should go here
   //d_cdf[threadPos] = d_bins[threadPos - 1] + d_cdf[threadPos - 1];
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
   int* d_bins;
   float* d_min, *d_max;
   checkCudaErrors(hipMalloc(&d_bins,   sizeof(int) * numBins));
   checkCudaErrors(hipMemset(d_bins, 0,   sizeof(int) * numBins));
   checkCudaErrors(hipMalloc(&d_min,   sizeof(float)));
   checkCudaErrors(hipMemcpy(d_min, &min_logLum, sizeof(float), hipMemcpyHostToDevice));
   checkCudaErrors(hipMalloc(&d_max,   sizeof(float)));
   checkCudaErrors(hipMemcpy(d_max, &max_logLum, sizeof(float), hipMemcpyHostToDevice));
   
   const size_t length = numRows * numCols;
   calc<<<numBins, length / numBins>>>(d_logLuminance, d_bins, d_cdf, d_min, d_max, numRows, numCols, numBins);
   checkCudaErrors(hipGetLastError());
   checkCudaErrors(hipMemcpy(&min_logLum, d_min, sizeof(float), hipMemcpyDeviceToHost));
   checkCudaErrors(hipMemcpy(&max_logLum, d_max, sizeof(float), hipMemcpyDeviceToHost));
   
   printf("%f\t%f\n", min_logLum, max_logLum);
   
   hipFree(d_bins);
   hipFree(d_min);
   hipFree(d_max);
}
